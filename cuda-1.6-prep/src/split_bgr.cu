#include "hip/hip_runtime.h"
#include "inc/cu_precomp.h"
#include "inc/cuda_img.h"

__global__
void cuda_split_bgr(CudaImg og, CudaImg b, CudaImg g, CudaImg r) {
    uint2 pos = {
        .x = blockDim.x * blockIdx.x + threadIdx.x,
        .y = blockDim.y * blockIdx.y + threadIdx.y,
    };
    
    if(pos.x >= og.size.x) return;
    if(pos.y >= og.size.y) return;

    uint tid = pos.y * og.size.x + pos.x;

    uchar3 color = og.p_uchar3[tid];

    b.p_uchar3[tid].x = color.x;
    g.p_uchar3[tid].y = color.y;
    r.p_uchar3[tid].z = color.z;
}

__host__
void cu_split_bgr(CudaImg og, CudaImg b, CudaImg g, CudaImg r) {
    dim3 block_dim(
        16, 16, 1
    );
    dim3 grid_dim(
        (og.size.x + block_dim.x - 1) / block_dim.x,
        (og.size.y + block_dim.y - 1) / block_dim.y,
        1
    );

    cuda_split_bgr<<<grid_dim, block_dim>>>(og, b, g, r);

    hipDeviceSynchronize();
}


__global__
void cuda_mirror(CudaImg og, uint8_t hor_vert) {
    uint2 pos = {
        .x = blockDim.x * blockIdx.x + threadIdx.x,
        .y = blockDim.y * blockIdx.y + threadIdx.y
    };

    uint tid = pos.y * og.size.x + pos.x;

    // hor
    uint mirror_tid;
    if (!hor_vert)
        mirror_tid = tid + og.size.x - 2 * pos.x - 1;
    else
        mirror_tid = (pos.y + og.size.y - 2 * pos.y) * og.size.x + pos.x - og.size.x;
    
    uchar3 temp = og.p_uchar3[tid];
    og.p_uchar3[tid] = og.p_uchar3[mirror_tid];
    og.p_uchar3[mirror_tid] = temp;
}

__host__
void cu_mirror(CudaImg og, uint8_t hor_vert) {
    dim3 bd;
    // 1024 = 2^10
    for(size_t i = 10; i > 0; i--) {
        bd.x = 1 << i;
        if((og.size.x / bd.x) % 2 == 0 && (og.size.x % bd.x) == 0) {
            break;
        }
    }

    for(size_t i = 10; i > 0; i--) {
        bd.y = 1 << i;
        if((og.size. y/ bd.y) % 2 == 0 && (og.size.y % bd.y) == 0) {
            break;
        }
    }
    printf("block_dim: x = %d, y = %d, z = %d\n", bd.x, bd.y, bd.z);

    // 0 = hor
    // 1 = vert
    
    dim3 gd(
        (og.size.x + bd.x - 1) / bd.x,
        (og.size.y + bd.y - 1) / bd.y,
        1
    );
    
    if (!hor_vert) {
        gd.x /= 2;
    } else {
        gd.y /= 2;
    }

    cuda_mirror<<< gd, bd >>>(og, hor_vert);
    
    hipDeviceSynchronize();
}