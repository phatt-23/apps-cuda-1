#include "hip/hip_runtime.h"
#include "module.h"

// qualifier <= kernel fn, runs on GPU 
__global__ 
void cuda_vec_scale(float* vec, size_t N, float scale) {
    // rule 1) No conditional jumps - i.e. if {} else {}, for {}, while {}
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid >= N) return; 

    vec[tid] *= scale;
}

// interface function <= runs on CPU
void cu_vec_scale(float* vec, size_t N, float scale) {
    // 1. allocate GPU memory
    float* cu_vec = nullptr;

    hipMalloc(&cu_vec, N * sizeof(float));
    
    // 2. copy from CPU to GPU memory
    hipMemcpy(cu_vec, vec, N * sizeof(float), hipMemcpyHostToDevice);

    // 3. calculate grid_size and block_size
    // * * * | * * *  
    // * * * | * * *  
    // * * * | * * * 
    // ------+-------
    // * * * | * * *  
    // * * * | * * *  
    // * * * | * * *  
    
    // vec[500], n = 500

    size_t block_size = 128; // count of threads in one block
    size_t grid_size = (N + block_size - 1) / block_size; // count of blocks in a grid


    printf("N = %ld, scale = %f, grid_size = %ld, block_size = %ld\n", 
        N, scale, grid_size, block_size);

    // 4. call kernel
    // fn<<< grid_size, block_size >>>(...)
    cuda_vec_scale<<< grid_size, block_size >>>(cu_vec, N, scale);

    // 5. copy back data from GPU to CPU 
    hipMemcpy(vec, cu_vec, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(cu_vec);
}

void cu_vec_scaleUM(float* vec, size_t N, float scale) {
    size_t block_size = 128; 
    size_t grid_size = (N + block_size - 1) / block_size; 

    cuda_vec_scale<<< grid_size, block_size >>>(vec, N, scale);
    
    hipDeviceSynchronize();
    
    return;
}