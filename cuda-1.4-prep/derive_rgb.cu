#include "hip/hip_runtime.h"
#include "module.h"

__global__
void cuda_drive_rgb(CudaImg img) {
    uint2 tid = {
        .x = blockDim.x * blockIdx.x + threadIdx.x,
        .y = blockDim.y * blockIdx.y + threadIdx.y,
    };
    if(tid.x >= img.size.x || tid.y >= img.size.y) return;

    size_t idx = tid.y * img.size.x + tid.x;
    uchar3 tmp = img.p_uchar3[idx];

    uchar3 bgr = {
        .x = tmp.y,
        .y = tmp.z,
        .z = tmp.x,
    };

    img.p_uchar3[idx] = bgr; 
}

__host__
void cu_derive_rgb(CudaImg img, uint2 block_size) {
    dim3 grid_size(
        (img.size.x + block_size.x - 1) / block_size.x,
        (img.size.y + block_size.y - 1) / block_size.y
    );

    dim3 _block_size(
        block_size.x, 
        block_size.y  
    );

    cuda_drive_rgb<<< grid_size, _block_size >>>(img);

    hipDeviceSynchronize();
}
