#include "hip/hip_runtime.h"
#include "inc/cu_precomp.h"
#include "inc/cu_module.h"

__global__
void cu_rotateS_img(CudaImg img, int direction) {

}

__host__
void cu_rotateS_img(CudaImg img, int direction) {
    dim3 bs(
        16, 16
    );

    dim3 gs(
        (bs.x + img.size.x - 1) / bs.x,
        (bs.y + img.size.y - 1) / bs.y
    );

    

    hipDeviceSynchronize();
}