#include "hip/hip_runtime.h"
#include "inc/cu_module.h"

__global__
void cuda_separate_rgb_elems(CudaImg og, CudaImg b, CudaImg g, CudaImg r) {
    uint2 _tid = {
        .x = blockDim.x * blockIdx.x + threadIdx.x,
        .y = blockDim.y * blockIdx.y + threadIdx.y,
    };

    if(og.size.y <= _tid.y) return;
    if(og.size.x <= _tid.x) return;

    size_t tid = _tid.y * og.size.x + _tid.x;
    uchar3 color = og.p_uchar3[tid];

    b.p_uchar3[tid].x = color.x;
    g.p_uchar3[tid].y = color.y;
    r.p_uchar3[tid].z = color.z;
}

__host__
void cu_separate_rgb_elems(CudaImg og, CudaImg b, CudaImg g, CudaImg r) {
    dim3 block_size( 16, 16 );
    dim3 grid_size( 
        (block_size.x + og.size.x - 1) / block_size.x,
        (block_size.y + og.size.y - 1) / block_size.y
    );

    cuda_separate_rgb_elems<<<grid_size, block_size>>>(og, b, g, r);

    hipDeviceSynchronize();
}

