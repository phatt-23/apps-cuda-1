#include "hip/hip_runtime.h"
#include "inc/cu_precomp.h"
#include "inc/cu_module.h"

__global__
void cuda_flip_img(CudaImg img, size_t hv) {
    uint2 _tid = {
        .x = blockDim.x * blockIdx.x + threadIdx.x,
        .y = blockDim.y * blockIdx.y + threadIdx.y,
    };

    // return early if outside the image
    if(_tid.x >= img.size.x || _tid.y >= img.size.y) return; 

    size_t tid = _tid.y * img.size.x + _tid.x;
    size_t f_tid = 0;
    if(hv == 0)
        f_tid = ((_tid.y + 1) * img.size.x - 1) - _tid.x;
    else
        f_tid = ((img.size.y - _tid.y - 1) * img.size.x) + _tid.x;

    uchar3 tmp = img.p_uchar3[tid];
    img.p_uchar3[tid] = img.p_uchar3[f_tid];
    img.p_uchar3[f_tid] = tmp;
}

__host__
void cu_flip_img(CudaImg img, size_t hv) {
    dim3 bs(
        16,
        16
    );
    // 256 = 16**2
    dim3 gs(
        (bs.x + img.size.x - 1) / bs.x,
        (bs.y + img.size.y - 1) / bs.y
    );

    printf("block_size: %d, %d, %d\n", bs.x, bs.y, bs.z);
    printf("grid_size:  %d, %d, %d\n", gs.x, gs.y, gs.z);

    if(hv == 0) 
        gs.x = (gs.x + 1) / 2; 
    else 
        gs.y = (gs.y + 1) / 2;

    printf("grid_size:  %d, %d, %d\n", gs.x, gs.y, gs.z);

    cuda_flip_img<<<gs, bs>>>(img, hv);

    hipDeviceSynchronize();
}
