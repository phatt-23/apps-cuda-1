#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include <hip/driver_types.h>
#include <stdio.h>

__global__ void thread_hierarchy() {
    int l_x = threadIdx.x + blockIdx.x * blockDim.x;
    int l_y = threadIdx.y + blockIdx.y * blockDim.y;
    printf("Block{%d,%d}[%d,%d] Thread{%d,%d}[%d,%d] [%d,%d]\n",
        gridDim.x, gridDim.y, blockIdx.x, blockIdx.y,
        blockDim.x, blockDim.y, threadIdx.x, threadIdx.y, l_x, l_y);
}

void cu_run_cuda(dim3 t_grid_size, dim3 t_block_size) {
    hipError_t l_cerr;
    
    thread_hierarchy<<< t_grid_size, t_block_size >>>();
    
    if((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void cuda_array_mul(float* array, size_t len, float factor)
{
	size_t i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= len) return;
	array[i] *= factor;
}

void cu_array_mul(float* array, size_t len, float factor)
{
	hipError_t err;
	int threads = 128;
	int blocks = (threads + len - 1) / threads;
	cuda_array_mul<<< blocks, threads >>>(array, len, factor);

	if((err = hipGetLastError()) != hipSuccess) {
		printf("ERROR: [%d] '%s'\n", __LINE__, hipGetErrorString(err));
	}

	hipDeviceSynchronize();
}

