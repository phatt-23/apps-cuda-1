#include "hip/hip_runtime.h"
#include "module.h"

// __global__ __host__ __device__

__global__ // callable from CPU
void cuda_toupper(char* str, size_t len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(tid >= len || str[tid] < 'a' || str[tid] > 'z') return;

    str[tid] &= 0xdf;
    // str[tid] -= 'a' - 'A';
}

__host__ 
void cu_toupper(char* str) {
    size_t block_size = 32;
    size_t grid_size = (strlen(str) + block_size - 1) / block_size;
    
    printf("INFO: gs = %ld, bs = %ld\n\n", grid_size, block_size);
    
    cuda_toupper<<< grid_size, block_size >>>(str, strlen(str));

    hipDeviceSynchronize();
}
